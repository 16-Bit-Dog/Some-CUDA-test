#include "hip/hip_runtime.h"
// this an example with an excess result to show it works with complex things

#include "hip/hip_runtime.h" 
#include "" 
#include <stdio.h> 
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/device_free.h> // place holder for testing
#include <thrust/device_malloc.h> // place holder for testing

#pragma hd_warning_disable // intellisense goes nuts with CUDA - thinks everything is an error

__global__ void Add(double* a, double* b) {

	int index = threadIdx.x + 2;

	b[index - 2] = (a[index + 1] - a[index]) / 2;

	b[index - 2] += (a[index] - a[index - 1]) / 2;

	// current problem is looping final to first element

}


extern "C" int a() {
	int numberOfVillage = 0;
	int temp = 0;

	scanf("%d", &numberOfVillage); // input int

	thrust::host_vector <double> position(numberOfVillage); 

	for (int i = 0; i < numberOfVillage; i++) {

		scanf("%d", &temp); 
		position[i] = temp;

	}

	thrust::sort(thrust::host, position.begin(), position.end()); 
	thrust::host_vector <double> distance(numberOfVillage - 3, 0);

	if (distance.size() == 0) { 
  
     distance.resize(1);


		distance[0] = (position[2] - position[1]) / 2;
		distance[0] += (position[1] - position[0]) / 2;

		printf("%.1f", distance[0]);


		return 0;
	}
	else {

		thrust::device_vector <double> devPosition = position;
		thrust::device_vector <double> devDistance = distance;
		double* miniPosition = thrust::raw_pointer_cast(&devPosition[0]); 
		double* miniDistance = thrust::raw_pointer_cast(&devDistance[0]); 
		int threadSize = (position.size() - 2);

		Add << < 1, threadSize >> > (miniPosition, miniDistance);


		thrust::sort(thrust::host, devDistance.begin(), devDistance.end()); 
		distance = devDistance; 

		printf("%.1f", distance[0]);

		return 0;
	}


	return 0;

}
