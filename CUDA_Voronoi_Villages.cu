#include "hip/hip_runtime.h"
// https://dmoj.ca/problem/ccc18s1

// also if you're using cuda 11.1 [with the thrust that ships with - 1.9.x] the program will not work as expected due to a bug --> use release to test/use 

// this program should work on cuda 3.x enabled devices (kelper architechture)? really depends on if you have thrust with the cuda version installed
// this version of thrust also has issues...

#include "hip/hip_runtime.h" // standart cuda include
#include "" // standart cuda include
#include <stdio.h> // for printf - since it is predictable and scanf since it is supported by cuda 11.0
//#include <iostream> // for cout due to it being quick to test
//
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <thrust/device_free.h> // place holder for testing
#include <thrust/device_malloc.h> // place holder for testing

#pragma hd_warning_disable // intellisense goes nuts with CUDA - thinks everything is an error


// I DID NOT use a class mainly because it is not really more or less efficent to a important level

__global__ void Add(double* a, double* b) {

	int index = threadIdx.x + 2;

		b[index - 2] = (a[index + 1] - a[index]) / 2;

		b[index - 2] += (a[index] - a[index - 1]) / 2;
	
	// current problem is looping final to first element

}


int main() {
	int numberOfVillage = 0;
	int temp = 0;
		
	scanf("%d", &numberOfVillage); // input int

	thrust::host_vector <double> position (numberOfVillage); // I start off with a host vector since I need a predictable manuverable vector which will be seen later
	
	//while (true) { std::cout << position.size(); }; // <-- test if size is working due to error reached
	
	for (int i = 0; i < numberOfVillage; i++) {

		scanf("%d", &temp); // since scanf() is a cpu read from IO stream, it is a REALLY BAD IDEA to push this data directly to a device vector (would write and read more)
		
		position[i] = temp; 
		
	}
		
	thrust::sort(thrust::host, position.begin(), position.end()); // sorting so I can get a predictable result --> done on CPU since when done on the gpu I was getting runtime issues [that seem like a bug with CUDA or thrust]


	thrust::host_vector <double> distance (numberOfVillage-3, 0);
		
	if (distance.size() == 0) { // seperate because the CPU doing this is WAY WAY faster than gpu since its only 4 read and 2 write (and then 1 more read and IO stream output from the cpu)
		distance.resize(1);


		distance[0] = (position[2] - position[1]) / 2;
		distance[0] += (position[1] - position[0]) / 2;
		
		printf("%.1f", distance[0]);
		

		return 0;
	}
	else {

		thrust::device_vector <double> devPosition = position;
		thrust::device_vector <double> devDistance = distance; // better to call 1 CPU to GPU call for position and distance rather than many per kernel that is runing parallel on the GPU 

		

		double* miniPosition = thrust::raw_pointer_cast(&devPosition[0]); // device vectors are unable to be passed through a kernel, so I made a pointer to my device vector

		double* miniDistance = thrust::raw_pointer_cast(&devDistance[0]); // device vectors are unable to be passed through a kernel, so I made a pointer to a device vector 

		int threadSize = (position.size() - 2); // 
		
//		Add <<< 1, (threadSize*32)/32 >>> (miniPosition, miniDistance);
		
		Add << < 1, threadSize >> > (miniPosition, miniDistance);
		
		
		thrust::sort(thrust::host, devDistance.begin(), devDistance.end()); // sorting on the gpu since that was the point of this practice I did more my self

		distance = devDistance; //set device vector to host vector since 1 read and write is sometimes faster (depends on mostly on memory bandwidth) than a read and write to IO stream


		printf("%.1f", distance[0]);

		return 0;
	}


	return 0;

}
